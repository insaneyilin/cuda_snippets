/* 
 * Copyright 2020 insaneyilin All Rights Reserved.
 * 
 * 
 */

#include "../common/common.h"

int main(int argc, char **argv) {
  hipDeviceProp_t prop;
  int dev_id = -1;

  CHECK_CUDA_ERROR(hipGetDevice(&dev_id));
  printf("ID of current CUDA device:  %d\n", dev_id);

  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 1;
  prop.minor = 3;
  CHECK_CUDA_ERROR(hipChooseDevice(&dev_id, &prop));
  printf("ID of CUDA device closest to revision 1.3:  %d\n", dev_id);

  CHECK_CUDA_ERROR(hipSetDevice(dev_id));

  return 0;
}

